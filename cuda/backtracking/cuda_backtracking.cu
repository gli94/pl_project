#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cuda_backtracking.cuh"
#include "CycleTimer.h"

#define N 9
#define BLOCK_SIZE 3
#define UNASSINED 0

__device__
void clearBitmap(bool *map, int size) {
    for (int i = 0; i < size; i++) {
        map[i] = false;
    }
}


/**
 * This device checks the entire board to see if it is valid.
 *
 * board: this is a N * N sized array that stores the board to check. Rows are stored contiguously,
 *        so to access row r and col c, use board[r * N + c]
 */
__device__
bool validBoard(const int *board) {
    bool seen[N];
    clearBitmap(seen, N);
    
    // check if rows are valid
    for (int i = 0; i < N; i++) {
        clearBitmap(seen, N);
        
        for (int j = 0; j < N; j++) {
            int val = board[i * N + j];
            
            if (val != 0) {
                if (seen[val - 1]) {
                    return false;
                } else {
                    seen[val - 1] = true;
                }
            }
        }
    }
    
    // check if columns are valid
    for (int j = 0; j < N; j++) {
        clearBitmap(seen, N);
        
        for (int i = 0; i < N; i++) {
            int val = board[i * N + j];
            
            if (val != 0) {
                if (seen[val - 1]) {
                    return false;
                } else {
                    seen[val - 1] = true;
                }
            }
        }
    }
    
    int n = BLOCK_SIZE;
    
    // finally check if the sub-boards are valid
    for (int ridx = 0; ridx < n; ridx++) {
        for (int cidx = 0; cidx < n; cidx++) {
            clearBitmap(seen, N);
            
            for (int i = 0; i < n; i++) {
                for (int j = 0; j < n; j++) {
                    int val = board[(ridx * n + i) * N + (cidx * n + j)];
                    
                    if (val != 0) {
                        if (seen[val - 1]) {
                            return false;
                        } else {
                            seen[val-1] = true;
                        }
                    }
                }
            }
        }
    }
    
    
    // if we get here, then the board is valid
    return true;
}

__device__
bool validBoard(const int *board, int changed) {
    
    int r = changed / 9;
    int c = changed % 9;
    
    // if changed is less than 0, then just default case
    if (changed < 0) {
        return validBoard(board);
    }
    
    if ((board[changed] < 1) || (board[changed] > 9)) {
        return false;
    }
    
    bool seen[N];
    clearBitmap(seen, N);
    
    // check if row is valid
    for (int i = 0; i < N; i++) {
        int val = board[r * N + i];
        
        if (val != 0) {
            if (seen[val - 1]) {
                return false;
            } else {
                seen[val - 1] = true;
            }
        }
    }
    
    // check if column is valid
    clearBitmap(seen, N);
    for (int j = 0; j < N; j++) {
        int val = board[j * N + c];
        
        if (val != 0) {
            if (seen[val - 1]) {
                return false;
            } else {
                seen[val - 1] = true;
            }
        }
    }
    
    int n = BLOCK_SIZE;
    // finally check if the sub-board is valid
    int ridx = r / n;
    int cidx = c / n;
    
    clearBitmap(seen, N);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int val = board[(ridx * n + i) * N + (cidx * n + j)];
            
            if (val != 0) {
                if (seen[val - 1]) {
                    return false;
                } else {
                    seen[val - 1] = true;
                }
            }
        }
    }
    
    // if we get here, then the board is valid
    return true;
}

__device__
bool checkrow(int *grid, int Num, int row)
{
 
    bool seen[N];
    
    for (int i = 0; i < Num; i++)
    {
        seen[i] = false;
    }
        
    for (int col = 0; col < Num; col++)
    {
        /*if ((grid[row * Num + col] == value) && col != c);
        {
            return false;
        }*/
        int val = grid[row * Num + col];
        if (val > 0)
        {
            if(seen[val-1])
            {
                return false;
            }
            else
            {
                seen[val-1] = true;
            }
        }
    }
    
    return true;
}

__device__
bool checkcol(int *grid, int Num, int col)
{
    bool seen[N];
    
    for (int i = 0; i < Num; i++)
    {
        seen[i] = false;
    }
    
    for (int row = 0; row < Num; row++)
    {
        /*if ((grid[row * Num + col] == value) && row != r)
        {
            return false;
        }*/
        int val = grid[row * Num + col];
        if (val > 0)
        {
            if(seen[val-1])
            {
                return false;
            }
            else
            {
                seen[val-1] = true;
            }
        }
    }
    
    return true;
}

__device__
bool checkbox(int *grid, int Num, int box_start_row, int box_start_col)
{
    
    bool seen[N];
    
    for (int i = 0; i < Num; i++)
    {
        seen[i] = false;
    }
    
    for (int row = box_start_row; row < box_start_row + BLOCK_SIZE; row++)
    {
        for (int col = box_start_col; col < box_start_col + BLOCK_SIZE; col++)
        {
            /*if ((grid[row * Num + col] == value) && (row != r) && (col != c))
            {
                return false;
            }*/
            int val = grid[row * Num + col];
            if (val > 0)
            {
                if(seen[val-1])
                {
                    return false;
                }
                else
                {
                    seen[val-1] = true;
                }
            }
            
        }
    }
    
    return true;
}

__device__
bool isvalid(int *grid, int Num, int row, int col)
{
    /*if ((grid[row * N + col] < 1) || (grid[row * N + col] > 9)) {
        return false;
    }*/
    
    if (checkrow(grid, Num, row) && checkcol(grid, Num, col) && checkbox(grid, Num, row - row % BLOCK_SIZE, col - col % BLOCK_SIZE)/* && (grid[row * Num + col] == UNASSINED)*/)
    {
        return true;
    }
    else
    {
        return false;
    }
}

__global__
void sudoku_backtrack( int *boards,
                       const int num_boards,
                       int *empty_spaces,
                       int *num_empty_spaces,
                       int *finished,
                       int *solved)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    int *currentBoard;
    int *currentEmptySpaces;
    int currentNumEmptySpaces;
    
    
    while((*finished == 0) && (index < num_boards))
    {
        
        int emptyIndex = 0;
        int row;
        int col;
        //int value = 0;
        
        currentBoard = boards + index * N * N;
        currentEmptySpaces = empty_spaces + index * N * N;
        currentNumEmptySpaces = num_empty_spaces[index];
        
        
        while ((emptyIndex >= 0) && (emptyIndex < currentNumEmptySpaces))
        {
            currentBoard[currentEmptySpaces[emptyIndex]]++;
            
            row = currentEmptySpaces[emptyIndex] / N;
            col = currentEmptySpaces[emptyIndex] % N;
 
            
            if(!isvalid(currentBoard, N, row, col))
            {
                if(currentBoard[currentEmptySpaces[emptyIndex]] >= 9)
                {
                    currentBoard[currentEmptySpaces[emptyIndex]] = 0;
                    emptyIndex--;
                }
            }
            else
            {
                //printf("Valid!\n");
               // printf("EmptyIndex = %d, EmptySpaces = %d \n", emptyIndex, currentNumEmptySpaces);
                //currentBoard[currentEmptySpaces[emptyIndex]] = value;
                //printf("Value filled in: %d\n", currentBoard[currentEmptySpaces[emptyIndex]]);
                //value = 0;
                emptyIndex++;
            }
        }
        
        if(emptyIndex == currentNumEmptySpaces)
        {
            *finished = 1;
            
            printf("Finished!\n");
            
            for (int i = 0; i < N * N; i++)
            {
                solved[i] = currentBoard[i];
            }
        }
        
        index += gridDim.x * blockDim.x;
    }
}

/*__global__
void sudoku_backtrack(int *boards,
                     const int numBoards,
                     int *emptySpaces,
                     int *numEmptySpaces,
                     int *finished,
                     int *solved) {
    
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    int *currentBoard;
    int *currentEmptySpaces;
    int currentNumEmptySpaces;
    
    
    while ((*finished == 0) && (index < numBoards)) {
        
        int emptyIndex = 0;
        
        currentBoard = boards + index * 81;
        currentEmptySpaces = emptySpaces + index * 81;
        currentNumEmptySpaces = numEmptySpaces[index];
        
        while ((emptyIndex >= 0) && (emptyIndex < currentNumEmptySpaces)) {
            
            currentBoard[currentEmptySpaces[emptyIndex]]++;
            
            if (!validBoard(currentBoard, currentEmptySpaces[emptyIndex])) {
                
                // if the board is invalid and we tried all numbers here already, backtrack
                // otherwise continue (it will just try the next number in the next iteration)
                if (currentBoard[currentEmptySpaces[emptyIndex]] >= 9) {
                    currentBoard[currentEmptySpaces[emptyIndex]] = 0;
                    emptyIndex--;
                }
            }
            // if valid board, move forward in algorithm
            else {
                emptyIndex++;
            }
            
        }
        
        if (emptyIndex == currentNumEmptySpaces) {
            // solved board found
            *finished = 1;
            
            // copy board to output
            for (int i = 0; i < N * N; i++) {
                solved[i] = currentBoard[i];
            }
        }
        
        index += gridDim.x * blockDim.x;
    }
}*/


void cuda_sudokuBacktrack (const int blocksPerGrid,
                           const int threadsPerBlock,
                           int * boards,
                           const int num_boards,
                           int *empty_spaces,
                           int *num_empty_spaces,
                           int *finished,
                           int *solved)
{
    sudoku_backtrack<<<blocksPerGrid, threadsPerBlock>>>(boards, num_boards, empty_spaces, num_empty_spaces, finished, solved);
}

/*__global__
void cudaBFSKernel (int *old_boards,
                    int *new_boards,
                    int total_boards,
                    int *board_index,
                    int *empty_spaces,
                    int *empty_space_count
                    )
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int *current_old_board;
    
    while (index < total_boards)
    {
        int found = 0;
        //current_old_board = old_boards + index * N * N;
        
        for (int i = index * N * N; (i < (index * N * N + N * N) && (found == 0)); i++)
        {
            if (old_boards[i] == UNASSINED)
            {
                found = 1;
                int row = (i - index * N * N) / N;
                int col = (i - index * N * N) % N;
                
                for (int attempt = 1; attempt <= N; attempt++)
                {
                    int works = 1;
                    
                    //if (!isvalid(current_old_board, N, row, col, attempt))
                    //{
                    //    works = 0;
                    //}
                    
                    for (int c = 0; c < N; c++) {
                        if (old_boards[row * N + c + N * N * index] == attempt) {
                            works = 0;
                        }
                    }
                    // column contraint, test various rows
                    for (int r = 0; r < N; r++) {
                        if (old_boards[r * N + col + N * N * index] == attempt) {
                            works = 0;
                        }
                    }
                    // box constraint
                    for (int r = n * (row / n); r < n; r++) {
                        for (int c = n * (col / n); c < n; c++) {
                            if (old_boards[r * N + c + N * N * index] == attempt) {
                                works = 0;
                            }
                        }
                    }
                    
                    if (works == 1)
                    {
                        int next_board_index = atomicAdd(board_index, 1);
                        int empty_index = 0;
                        
                        for (int r = 0; r < N; r++)
                        {
                            for (int c = 0; c < N; c++)
                            {
                                new_boards[next_board_index * N * N + r * N + c] = old_boards[index * N * N + r * N + c];
                                if (old_boards[index * N * N + r * N + c] == 0 && (r != row || c != col))
                                {
                                    empty_spaces[empty_index + next_board_index * N * N] = r * N + c;
                                    empty_index++;
                                }
                            }
                        }
                        
                        empty_space_count[next_board_index] = empty_index;
                        new_boards[next_board_index * N * N + row * N + col] = attempt;
                    }
                }
            }
        }
        
        index += blockDim.x * gridDim.x;
    }
}*/

__global__
void
cudaBFSKernel(int *old_boards,
              int *new_boards,
              int total_boards,
              int *board_index,
              int *empty_spaces,
              int *empty_space_count) {
    
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // board_index must start at zero
    
    while (index < total_boards) {
        // find the next empty spot
        int found = 0;
        
        for (int i = (index * N * N); (i < (index * N * N) + N * N) && (found == 0); i++) {
            // found a open spot
            if (old_boards[i] == 0) {
                found = 1;
                // get the correct row and column shits
                int temp = i - N * N * index;
                int row = temp / N;
                int col = temp % N;
                
                // figure out which numbers work here
                for (int attempt = 1; attempt <= N; attempt++) {
                    int works = 1;
                    // row constraint, test various columns
                    for (int c = 0; c < N; c++) {
                        if (old_boards[row * N + c + N * N * index] == attempt) {
                            works = 0;
                        }
                    }
                    // column contraint, test various rows
                    for (int r = 0; r < N; r++) {
                        if (old_boards[r * N + col + N * N * index] == attempt) {
                            works = 0;
                        }
                    }
                    // box constraint
                    for (int r = BLOCK_SIZE * (row / BLOCK_SIZE); r < BLOCK_SIZE * (row / BLOCK_SIZE) + BLOCK_SIZE; r++) {
                        for (int c = BLOCK_SIZE * (col / BLOCK_SIZE); c < BLOCK_SIZE * (col / BLOCK_SIZE) + BLOCK_SIZE; c++) {
                            if (old_boards[r * N + c + N * N * index] == attempt) {
                                works = 0;
                            }
                        }
                    }
                    if (works == 1) {
                        // copy the whole board
                        
                        int next_board_index = atomicAdd(board_index, 1);
                        int empty_index = 0;
                        for (int r = 0; r < 9; r++) {
                            for (int c = 0; c < 9; c++) {
                                new_boards[next_board_index * 81 + r * 9 + c] = old_boards[index * 81 + r * 9 + c];
                                if (old_boards[index * 81 + r * 9 + c] == 0 && (r != row || c != col)) {
                                    empty_spaces[empty_index + 81 * next_board_index] = r * 9 + c;
                                    
                                    empty_index++;
                                }
                            }
                        }
                        empty_space_count[next_board_index] = empty_index;
                        new_boards[next_board_index * 81 + row * 9 + col] = attempt;
                        
                    }
                }
            }
        }
        
        index += blockDim.x * gridDim.x;
    }
}

void callBFSKernel( const int blocksPerGrid,
                    const int threadsPerBlock,
                   int *old_boards,
                   int *new_boards,
                   int total_boards,
                   int *board_index,
                   int *empty_spaces,
                   int *empty_space_count)
{
    cudaBFSKernel<<<blocksPerGrid, threadsPerBlock>>>(old_boards, new_boards, total_boards, board_index, empty_spaces, empty_space_count);
}

void cuda_Backtrack(int * board, int * solved)
{
    int blocksPerGrid = 1024;
    int threadsPerBlock = 256;
 
    int *old_boards;
    int *new_boards;
    int *empty_spaces;
    int *empty_space_count;
    int *board_index;
    
    int sk = pow(2, 26);
    
    hipMalloc(&empty_spaces, sk * sizeof(int));
    hipMalloc(&empty_space_count, ((sk / 81) + 1) * sizeof(int));
    hipMalloc(&old_boards, sk * sizeof(int));
    hipMalloc(&new_boards, sk * sizeof(int));
    hipMalloc(&board_index, sizeof(int));
    
    int total_boards = 1;
    
    hipMemset(board_index, 0, sizeof(int));
    hipMemset(new_boards, 0, sk * sizeof(int));
    hipMemset(old_boards, 0, sk * sizeof(int));
    
    hipMemcpy(old_boards, board, N * N * sizeof(int), hipMemcpyHostToDevice);
    
    double startGPUTime = CycleTimer::currentSeconds();
    callBFSKernel(blocksPerGrid, threadsPerBlock, old_boards, new_boards, total_boards, board_index, empty_spaces, empty_space_count);
    
    
    int host_count;
    
    int iterations = 18;
    
    for (int i=0; i<iterations; i++)
    {
        hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost);
        printf("total boards after an iteration %d: %d\n", i, host_count);
        hipMemset(board_index, 0, sizeof(int));
        
        if((i % 2) == 0)
        {
            callBFSKernel(blocksPerGrid, threadsPerBlock, new_boards, old_boards, host_count, board_index, empty_spaces, empty_space_count);
        }
        else
        {
            callBFSKernel(blocksPerGrid, threadsPerBlock, old_boards, new_boards, host_count, board_index, empty_spaces, empty_space_count);
        }
    }
    
    hipMemcpy(&host_count, board_index, sizeof(int), hipMemcpyDeviceToHost);
    printf("new number of boards retrieved is %d\n", host_count);
    
    int *dev_finished;
    int *dev_solved;
    
    hipMalloc(&dev_finished, sizeof(int));
    hipMalloc(&dev_solved, N * N * sizeof(int));
    
    hipMemset(dev_finished, 0, sizeof(int));
    hipMemcpy(dev_solved, board, N * N * sizeof(int), hipMemcpyHostToDevice);
    
    if((iterations % 2) == 1)
    {
        new_boards = old_boards;
    }
    
    cuda_sudokuBacktrack(blocksPerGrid, threadsPerBlock, new_boards, host_count, empty_spaces, empty_space_count, dev_finished, dev_solved);
    double endGPUTime = CycleTimer::currentSeconds();
    double timeKernel = endGPUTime - startGPUTime;
    
    printf("Execution time: %lfs\n", timeKernel);
    
    //int *solved = new int[N * N];
    //memset(solved, 0, N * N * sizeof(int));
    hipMemcpy(solved, dev_solved, N * N * sizeof(int), hipMemcpyDeviceToHost);
    double endGPUTime2 = CycleTimer::currentSeconds();
    
    printf("Memcpy time: %lfs\n", endGPUTime2-endGPUTime);
    //printBoard(solved);
    
    //delete[] board;
    //delete[] solved;
    
    hipFree(empty_spaces);
    hipFree(empty_space_count);
    hipFree(new_boards);
    hipFree(old_boards);
    hipFree(board_index);
    
    hipFree(dev_finished);
    hipFree(dev_solved);
}
